#include "hip/hip_runtime.h"
#include "helper.cuh"

// the utils function of print array
void PrintArrayAPI(float* Arr, unsigned n)
{
    for (unsigned i = 0; i != n; i++){
		printf("%f ", float(Arr[i]));
	}
	printf("\n");
}

// the kernel function of computing back propagation in Softmax layer
__global__ void SoftmaxLossBackprop(
	const float* result, float* error,
	unsigned const classCount, unsigned const batch_size,
	const unsigned* label_ptr)
{
	unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
    // using one single line to avoid the divergence
    error[idx] = 1.0f * (threadIdx.x == label_ptr[blockIdx.x]) - result[idx];
}

// the utils function of computing back propagation in Softmax layer
void SoftmaxLossBackpropAPI(
	const float* result, float* error,
	unsigned const classCount, unsigned const batch_size,
	const unsigned* label_ptr)
{
	SoftmaxLossBackprop << < batch_size, classCount >> > (result, error, classCount, batch_size, label_ptr);
	hipDeviceSynchronize();
}





